#include "NvInfer.h"
#include "NvOnnxParser.h" 
#include "logger.h"
#include "common.h"
#include "buffers.h"
#include "cassert"
#include "utils/config.h"
#include "utils/preprocess.h"
#include "utils/types.h"

// 定义校准数据读取器, 最大最小值校准
// 如果要用熵校准entropy的话改为：IInt8EntropyCalibrator2
class CalibrationDataReader : public nvinfer1::IInt8MinMaxCalibrator
{
private:
    std::string mDataDir;
    std::string mCacheFileName;
    std::vector<std::string> mFileNames;
    int mBatchSize;
    nvinfer1::Dims mInputDims;
    int mInputCount;
    float *mDeviceBatchData { nullptr };
    int mBatchCount;
    int mImgSize;
    int mCurBatch{0};
    std::vector<char> mCalibrationCache;

private:
    void load_dataClassFile(const std::string& filepath)
    {
        std::ifstream ifile(filepath);
        std::string Line;
        while (std::getline(ifile, Line))
        {
            sample::gLogInfo << Line << std::endl;
            mFileNames.push_back(Line);
        }
        mBatchCount = mFileNames.size() / mBatchSize;
        std::cout << "CalibrationDataReader: " << mFileNames.size() 
                  << " images, " << mBatchCount << " batches." << std::endl;
    }

public:
    // 构造函数需要传递的参数包括数据目录、数据列表、BatchSize。
    // 通常会根据模型的需求，初始化输入张量的维度和大小，并在设备上分配相应的内存。
    CalibrationDataReader(const std::string& dataDir, const std::string& filepath, int batchSize = 1)
        : mDataDir(dataDir), mCacheFileName("weights/calibration.cache"),
          mBatchSize(batchSize), mImgSize(kInputH * kInputW)
    {
        mInputDims = {1, 3, kInputH, kInputW};
        mInputCount = mBatchSize * samplesCommon::volume(mInputDims);
        cuda_preprocess_init(mImgSize);
        hipMalloc(&mDeviceBatchData, kInputH * kInputW * 3 * sizeof(float));
        load_dataClassFile(filepath);
    }

    int32_t getBatchSize() const noexcept override
    {
        return mBatchSize;
    }

    bool getBatch(void* bindings[], const char *names[], int nbBindings) noexcept override
    {
        if (mCurBatch + 1 > mBatchCount)
        {
            return false;
        }
        int offset = kInputW * kInputH * 3 * sizeof(float);
        for (int i = 0; i < mBatchSize; i++)
        {
            int idx = mCurBatch * mBatchSize + i;
            std::string filename = mDataDir + "/" + mFileNames[idx];
            cv::Mat image = cv::imread(filename);
            int new_img_size = image.cols * image.rows;
            if (new_img_size > mImgSize)
            {
                mImgSize = new_img_size;
                cuda_preprocess_destroy();
                cuda_preprocess_init(mImgSize);
            }
            process_input_gpu(image, mDeviceBatchData + i * offset);
        }
        for (int i = 0; i < nbBindings; i++)
        {
            if (!strcmp(names[i], kInputTensorName))
            {
                bindings[i] = mDeviceBatchData + i * offset;
            }
        }
        mCurBatch++;
        return true;
    }

    const void* readCalibrationCache(std::size_t& length) noexcept override
    {
        mCalibrationCache.clear();

        std::ifstream input(mCacheFileName, std::ios::binary);
        input >> std::noskipws;
        if (input.good())
        {
            std::copy(std::istream_iterator<char>(input), std::istream_iterator<char>(),
                      std::back_inserter(mCalibrationCache));
        }
        length = mCalibrationCache.size();
        return length ? mCalibrationCache.data() : nullptr;
    }

    void writeCalibrationCache(const void *cache, std::size_t length) noexcept override
    {
        std::ofstream output(mCacheFileName, std::ios::binary);
        output.write(reinterpret_cast<const char*>(cache), length);
    }
};


int main(int argc, char** argv)
{
    if (argc != 4)
    {
        std::cerr << "请输入onnx文件位置: ./build/[onnx_file] [calib_dir] [calib_list_file]" << std::endl;
        return -1;
    }
    // 命令行获取onnx文件路径、校准数据集路径、校准数据集列表文件
    char* onnx_file = argv[1];
    char* calib_dir = argv[2];
    char* calib_list_file = argv[3];
    // ========== 1. 创建builder：创建优化的执行引擎（ICudaEngine）的关键工具 ==========
    // 在几乎所有使用TensorRT的场合都会使用到IBuilder
    // 只要TensorRT来进行优化和部署，都需要先创建和使用IBuilder。
    std::unique_ptr<nvinfer1::IBuilder> builder = std::unique_ptr<nvinfer1::IBuilder>(nvinfer1::createInferBuilder(sample::gLogger.getTRTLogger()));
    if (!builder)
    {
        std::cerr << "Failed to create build" << std::endl;
        return -1;
    } 
    std::cout << "Successfully to create builder!!" << std::endl;

    // ========== 2. 创建network：builder--->network ==========
    // 设置batch, 数据输入的批次量大小
    // 显性设置batch
    const unsigned int explicitBatch = 1U << static_cast<uint32_t>(nvinfer1::NetworkDefinitionCreationFlag::kEXPLICIT_BATCH);
    std::unique_ptr<nvinfer1::INetworkDefinition> network = std::unique_ptr<nvinfer1::INetworkDefinition>(builder->createNetworkV2(explicitBatch));
    if (!network)
    {
        std::cout << "Failed to create network" << std::endl;
        return -1;
    }

    // 创建onnxparser，用于解析onnx文件
    std::unique_ptr<nvonnxparser::IParser> parser = std::unique_ptr<nvonnxparser::IParser>(nvonnxparser::createParser(*network, sample::gLogger.getTRTLogger()));
    // 调用onnxparser的parseFromFile方法解析onnx文件
    bool parsed = parser->parseFromFile(onnx_file, static_cast<int>(sample::gLogger.getReportableSeverity()));
    if (!parsed)
    {
        std::cerr << "Failed to parse onnx file!!" << std::endl;
        return -1;
    }
    // 配置网络参数
    // 我们需要告诉tensorrt我们最终运行时，输入图像的范围，batch size的范围。这样tensorrt才能对应为我们进行模型构建与优化。
    nvinfer1::ITensor* input = network->getInput(0); // 获取了网络的第一个输入节点。
    nvinfer1::IOptimizationProfile* profile = builder->createOptimizationProfile(); // 创建了一个优化配置文件。
    // 网络的输入节点就是模型的输入层，它接收模型的输入数据。
    // 在 TensorRT 中，优化配置文件（Optimization Profile）用于描述模型的输入尺寸和动态尺寸范围。
    // 通过优化配置文件，可以告诉 TensorRT 输入数据的可能尺寸范围，使其可以创建一个适应各种输入尺寸的优化后的模型。

    // 设置最小尺寸
    profile->setDimensions(input->getName(), nvinfer1::OptProfileSelector::kMIN, nvinfer1::Dims4(1, 3, 640, 640));
    // 设置最优尺寸
    profile->setDimensions(input->getName(), nvinfer1::OptProfileSelector::kOPT, nvinfer1::Dims4(1, 3, 640, 640));
    // 设置最大尺寸
    profile->setDimensions(input->getName(), nvinfer1::OptProfileSelector::kMAX, nvinfer1::Dims4(1, 3, 640, 640));

    // ========== 3. 创建config配置：builder--->config ==========
    // 配置解析器
    std::unique_ptr<nvinfer1::IBuilderConfig> config = std::unique_ptr<nvinfer1::IBuilderConfig>(builder->createBuilderConfig());
    if (!config)
    {
        std::cout << "Failed to create config" << std::endl;
        return -1;
    }
    // 添加之前创建的优化配置文件（profile）到配置对象（config）中
    // 优化配置文件（profile）包含了输入节点尺寸的设置，这些设置会在模型优化时被使用。
    config->addOptimizationProfile(profile);
    // 设置精度
    if (!builder->platformHasFastInt8())
    {
        sample::gLogInfo << "设备不支持int8，本次将默认使用int16" << std::endl;
        config->setFlag(nvinfer1::BuilderFlag::kFP16);
    }
    else {
        sample::gLogInfo << "设备支持int8，本次将使用int8量化" << std::endl;
        auto calibrator = new CalibrationDataReader(calib_dir, calib_list_file);
        config->setFlag(nvinfer1::BuilderFlag::kINT8);
        config->setInt8Calibrator(calibrator);
    }

    // config->setFlag(nvinfer1::BuilderFlag::kFP16);
    builder->setMaxBatchSize(1);
    config->setMemoryPoolLimit(nvinfer1::MemoryPoolType::kWORKSPACE, 1 << 30);

    // 创建流，用于设置profile
    auto profileStream = samplesCommon::makeCudaStream();
    if (!profileStream)
    {
        std::cerr << "Failed to create CUDA profileStream File" << std::endl;
        return -1;
    }
    config->setProfileStream(*profileStream);

    // ========== 5. 序列化保存engine ==========
    // 使用之前创建并配置的 builder、network 和 config 对象来构建并序列化一个优化过的模型。
    std::unique_ptr<nvinfer1::IHostMemory> plan = std::unique_ptr<nvinfer1::IHostMemory>(builder->buildSerializedNetwork(*network, *config));
    std::ofstream engine_file("./weights/best.engine", std::ios::binary);
    assert(engine_file.is_open() && "Failed to open engine file");
    engine_file.write((char *)plan->data(), plan->size());
    engine_file.close();

    // ========== 6. 释放资源 ==========
    sample::gLogInfo << "Engine build success!" << std::endl;
    return 0;
}