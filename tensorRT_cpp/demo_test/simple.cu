
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c, int N)
{
    // 获取本线程的索引，blockIdx 指的是线程块的索引，blockDim 指的是线程块的大小，threadIdx 指的是本线程块中的线程索引
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("tid: %d blockIdx.x: %d blockDim.x: %d threadIdx.x: %d \n", tid, blockIdx.x, blockDim.x, threadIdx.x);
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

int main(int argc, char **argv)
{
    // 检查命令行参数
    if (argc != 2)
    {
        fprintf(stderr, "Usage: ./simple <N>");
    }
    int N = std::atoi(argv[1]);
    int a[N], b[N], c[N], c_from_gpu[N];
    int *dev_a, *dev_b, *dev_c;

    // 在设备端分配内存
    hipMalloc((void **)&dev_a, N * sizeof(int));
    hipMalloc((void **)&dev_b, N * sizeof(int));
    hipMalloc((void **)&dev_c, N * sizeof(int));

    // 初始化数组
    for (int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = i;
    }

    // 统计CPU上运行时间
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for (int i = 0; i < N; i++)
    {
        c[i] = a[i] + b[i];
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);
    printf("Time spent on CPU: %f ms\n", time);

    // 将数据从主机端复制到设备端
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    // 调用kernel函数，在GPU上运行并发计算
    // 一个线程块包含256个线程
    int n_threads = 256;
    // 计算线程块的数量
    int n_blocks = std::ceil(N * 1.0f / n_threads);

    // 统计时间
    hipEventRecord(start, 0);
    // 调用kernel函数，传递线程块数量和大小
    add<<<n_blocks, n_threads>>>(dev_a, dev_b, dev_c, N); 
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Time spent on GPU: %f ms\n", time);

    // 将数据从设备端复制到主机端
    hipMemcpy(c_from_gpu, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // 检查结果是否一致
    for (int i = 0; i < N; i++)
    {
        if (c[i] != c_from_gpu[i])
        {
            printf("Error: inconsistent results!\n");
        }
    }

    // 释放设备端内存
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
